#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#define N 5
#define  INF 9999

int costMatrixGraph[N][N] = {
	{ 0, 10, INF, 5, INF },
	{ INF, 1, 6, INF, 7 },
	{ INF, INF, 0, 9, INF },
	{ INF, 9, INF, 0, INF },
	{ INF, INF, INF, 3, 0 }
};

__global__ void RoyFloyd(int costMatrixGraph[N][N], int k)
{

	int i = threadIdx.x;
	int j = threadIdx.y;

	if (costMatrixGraph[i][k] + costMatrixGraph[k][j] < costMatrixGraph[i][j]) {
		costMatrixGraph[i][j] = costMatrixGraph[i][k] + costMatrixGraph[k][j];
	}
}

int main()
{
	int *matrix;
	int* d_k;
	int numBlocks = 1;

	hipMalloc(&matrix, N*N * sizeof(int));
	hipMemcpy(matrix, costMatrixGraph, N * N * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc(&d_k, sizeof(int));
	for (int k = 0; k < N; ++k)
	{
		hipMemcpy(d_k, &k, sizeof(int), hipMemcpyHostToDevice);
		dim3 threadsPerBlock(N, N);
		RoyFloyd<<<numBlocks, threadsPerBlock>>>(matrix, k);
	}
	hipMemcpy(costMatrixGraph, matrix, N * N * sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; ++j)
		{
			if (costMatrixGraph[i][j] == INF)
				std::cout << "inf ,";
			else
				std::cout << costMatrixGraph[i][j] << ", ";
		}
		std::cout << std::endl;
	}

	hipFree(costMatrixGraph);
	hipFree(matrix);
	system("pause");
	return 0;

}
